#include "hip/hip_runtime.h"
/*
 * spmm_csc_driver.cu
 * Copyright (C) 2020
 *  Aravind SUKUMARAN RAJAM (asr) <aravind_sr@outlook.com>
 *
 * Distributed under terms of the GNU LGPL3 license.
 */

#include "mm_helper.hpp"
#include "sparse_representation.hpp"
#include <iostream>
#define BLK_SIZE 32


void check_dmat(double* a, double *b, unsigned int n, unsigned int K, bool quit_on_err = true ) {
    for (unsigned int i = 0; i < n; ++i) {
        for (unsigned int k = 0; k < K; ++k) {
            if(std::abs(a[i * K + k] - b[i * K + k]) > 1e-1) {
                std::cerr << "Possible error at " << i << std::endl;

                if(quit_on_err) {
                    exit(-1);
                }
            }
        }
    }
    if(quit_on_err)
        std::cout << "Verification succeeded\n";
    else
        std::cout << "Check error messages to see if verification succeeded. (No error msg == success)\n";
}

static unsigned int g_seed = 0X4B1D;
inline int fastrand() {
    g_seed = (214013 * g_seed + 2531011);
    return (g_seed >> 16) & 0x7FFF;
}

void init_dmat(double *a, unsigned int n, unsigned int K, double offset) {
    for (unsigned int i = 0; i < n; ++i) {
        for (unsigned int k = 0; k < K; ++k) {
            a[i * K + k]  = i * K + k + offset;
            //a[i * K + j]  = fastrand() + offset;
        }
    }
}

void print_dmat(double *a, unsigned int n, unsigned int K) {
    for (unsigned int i = 0; i < n; ++i) {
        for (unsigned int j = 0; j < K; ++j) {
            std::cout << a[i * K + j]   << ' ';
        }
        std::cout << '\n';
    }
}

void host_csc_spmm(CSC mat, double * dmat_in, double * dmat_out, unsigned int K) {
    for (unsigned int r = 0; r < mat.nrows; ++r) {
        for (unsigned int k = 0; k < K; ++k) {
            dmat_out[r * K + k] = 0;
        }
    }
    for (unsigned int c = 0; c < mat.ncols; ++c) {
        unsigned int col_start = mat.col_indx[c];
        unsigned int col_end = mat.col_indx[c + 1];

        for (unsigned int r = col_start; r < col_end; ++r) {
            unsigned int row_id = mat.row_id[r];
            double val = mat.values[r];

            for (unsigned int k = 0; k < K; ++k) {
                dmat_out[row_id * K + k] += val * dmat_in[c * K + k];
            }
        }

    }
}

__global__ void dev_csc_spmm(double *values, int *row_id, int *col_indx, int nnz, int ncols, int nrows, int K, const double *D, double *O){
    const int row = blockIdx.y * blockDim.y + threadIdx.y;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < K && col < nrows) {
       double dotProduct = 0;
       const int col_start = col_indx[col];
       const int col_end = col_indx[col+1];
       

       for (int element = col_start; element < col_end; ++element) {

	  dotProduct += values[element] * D[K * row_id[element] + row];
        
       }
    
       O[col * K + row] = dotProduct;
    
    }

}



int main(int argc, char *argv[]) {
    if(argc < 3) {
        std::cerr << "usage ./exec inputfile K  " << std::endl;
        exit(-1);
    }

    unsigned int K = std::atoi(argv[2]);
    CSC mat = read_matrix_market_to_CSC(argv[1]);
    std::cout << mat.nrows << ' ' << mat.ncols << ' ' << mat.nnz << ' ' << K << '\n';

    double *dmat_in = (double*)malloc(mat.ncols * K  * sizeof(double));
    double *dmat_out = (double*)malloc(mat.nrows * K * sizeof(double));

    init_dmat(dmat_in, mat.ncols, K, 1.0);
    //print_dmat(dmat_in, mat.ncols, K);

    host_csc_spmm(mat, dmat_in, dmat_out, K);
    //device array pointers
    double *d_values;
    int *d_row_id;
    int *d_col_indx;
    double *d_dmat_in;
    double *d_dmat_out;


    hipMalloc(&d_values, sizeof(double)* mat.nnz);
    hipMalloc(&d_row_id, sizeof(int) * (mat.nnz));
    hipMalloc(&d_col_indx, sizeof(int)* mat.ncols+1);
    hipMalloc(&d_dmat_in, sizeof(double)* K * mat.ncols);
    hipMalloc(&d_dmat_out, sizeof(double)* K * (mat.nrows));

    //----------- Begin kernel call for SpMM_CSR ------------
    float time_ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    //cudamemcopy functions
    hipMemcpy(d_values, mat.values, sizeof(double) * mat.nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_col_indx, mat.col_indx, sizeof(int) * mat.ncols+1, hipMemcpyHostToDevice);
    hipMemcpy(d_row_id, mat.row_id, sizeof(int) * (mat.nnz), hipMemcpyHostToDevice);
    hipMemcpy(d_dmat_in, dmat_in, sizeof(double)*K*mat.ncols, hipMemcpyHostToDevice);
    //hipMemcpy(d_dmat_out, dmat_out, sizeof(double)*K*(mat.nrows), hipMemcpyHostToDevice);

    //define blk and grid size
    dim3 threads(BLK_SIZE, BLK_SIZE);
    dim3 grid((int) ceil((float) K/BLK_SIZE), (int) ceil((float) mat.ncols/BLK_SIZE));


    //call gpu kernel
    dev_csc_spmm<<<grid, threads>>>(d_values, d_row_id, d_col_indx, mat.nnz, mat.ncols, mat.nrows, K, d_dmat_in, d_dmat_out);
    hipDeviceSynchronize();

    //cudamemcopy gpu result from device to host
    double *gpu_result = (double*)malloc((mat.nrows) * K  * sizeof(double));

    hipMemcpy(gpu_result, d_dmat_out, sizeof(double)*K*(mat.nrows), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_ms, start, stop);

    //compute GFLOPS
    double gflop = abs(((2 * K * mat.nnz)/1e9));
    double op_time_s = time_ms * 1e-3;
    double gflops = gflop/op_time_s;

    printf("Kernel time :  %f ms \n", time_ms);
    printf("GFLOPS :  %f \n", gflops);
   
    /*
    for(int i =0; i<mat.nrows*K; i++){
        printf("point: %d, gpu: %lf, cpu: %lf \n", i, gpu_result[i], dmat_out[i]);
    }
    */
   

    //std::cout << "replace one argument to the below function with the values from gpu " << std::endl;
    check_dmat(dmat_out, gpu_result, mat.nrows, K);

    //print_dmat(dmat_out, mat.nrows, K);


    free(mat.col_indx);
    free(mat.row_id);
    free(mat.values);
    free(gpu_result);

    hipFree(d_values);
    hipFree(d_col_indx);
    hipFree(d_row_id);
    hipFree(d_dmat_in);
    hipFree(d_dmat_out);
    
    return 0;
}
